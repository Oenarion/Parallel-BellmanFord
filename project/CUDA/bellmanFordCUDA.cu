#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MY_INFINITY 99999
#define BLK_DIM 256

//struct for the edges of the graph
struct Edge {
    int u;  //start vertex of the edge
    int v;  //end vertex of the edge
    int w;  //weight of the edge (u,v)
};

//Graph - it consists of edges
struct Graph {
    int V;        //total number of vertices in the graph
    int E;        //total number of edges in the graph
    struct Edge *edge;  //array of edges
};

__global__ void sequentialFillDistanceArray(int *d, int tV){
    for(int i=0; i<tV; i++){
        d[i] = MY_INFINITY;
    }
    d[0] = 0;
}

__global__ void sequentialRelaxationStep(struct Edge *edge, int tE, int *d){
    for(int i = 0; i < tE; i++){
        int u = edge[i].u;
        int v = edge[i].v;
        int w = edge[i].w;
        if (d[u] != MY_INFINITY && d[v] > d[u] + w) {
            d[v] = d[u] + w;
        }
    }
}

__global__ void sequentialCheckNegativeCycles(struct Edge *edge, int tE, int *d){
    for(int i = 0; i < tE; i++){
        int u = edge[i].u;
        int v = edge[i].v;
        int w = edge[i].w;
        if (d[u] != MY_INFINITY && d[v] > d[u] + w) {
            printf("Negative cycle found, no solution possible!\n");
        }
    }
}

__global__ void fillDistanceArray(int *d, int tV){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < tV) {
        d[i] = MY_INFINITY;
    }
    else if(i == 0){
        d[0] = 0;
    }
    
}

__global__ void relaxationStep(struct Edge *edge, int tE, int *d){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < tE){
        int u = edge[i].u;
        int v = edge[i].v;
        int w = edge[i].w;
        if (d[u] != MY_INFINITY && d[v] > d[u] + w) {
            atomicMin(&d[v], d[u] + w);
        }
    }
}

__global__ void checkNegativeCycles(struct Edge *edge, int tE, int *d){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < tE){
        int u = edge[i].u;
        int v = edge[i].v;
        int w = edge[i].w;
        if (d[u] != MY_INFINITY && d[v] > d[u] + w) {
            printf("Negative cycle found, no solution possible!\n");
        }
    }
}

void bellmanford(struct Graph *g, int source, int mode){
    int tV = g->V;
    int tE = g->E;
    int *d;
    hipMalloc(&d, tV * sizeof(int));

    // Allocate memory for edges on device
    // If we create a graph and use memcopy we are not copying the edges themselves
    // so to avoid this issue let's just copy the edges... (lost 2h fixing this)

    struct Edge *edge_dev;
    hipMalloc(&edge_dev, g->E * sizeof(struct Edge));

    // Copy edges from host to device
    hipMemcpy(edge_dev, g->edge, g->E * sizeof(struct Edge), hipMemcpyHostToDevice);

    //fill d
    if(mode == 0){
        fillDistanceArray<<<(tV + BLK_DIM - 1)/BLK_DIM, BLK_DIM>>>(d,tV);
        hipDeviceSynchronize();
    }
    else{
        sequentialFillDistanceArray<<<1,1>>>(d,tV);
    }

    // Relax edges for tV - 1 iterations
    for (int k = 1; k <= tV - 1; k++) {
        if(mode == 0){
            relaxationStep<<<(tE + BLK_DIM - 1)/BLK_DIM,BLK_DIM>>>(edge_dev, tE, d);
        }
        else{
            sequentialRelaxationStep<<<1,1>>>(edge_dev, tE, d);
        }
    }
    hipDeviceSynchronize();
    
    //check negative cycles
    if(mode == 0){
        checkNegativeCycles<<<(tE + BLK_DIM - 1)/BLK_DIM,BLK_DIM>>>(edge_dev, tE, d);
        hipDeviceSynchronize();
    }
    else{
        sequentialCheckNegativeCycles<<<1,1>>>(edge_dev, tE, d);
    }

    // Free device memory
    hipFree(d);
    hipFree(edge_dev);
}


int main(int argc, char* argv[]) {

    FILE *file;
    hipEvent_t start, stop;
    int numVertices = -1;
    int numEdges = -1;
    float elapsed_time;

    if (argc < 2) {
        printf("Usage: %s <filename>\n", argv[0]);
        return 1;
    }

    //opening the file
    file = fopen(argv[1], "r");

    if (file == NULL) {
        printf("Could not open the file.\n");
        return 1;
    }

    int mode = 1;

    if(argc < 3){
        printf("Default mode: sequential\n");
    }
    else if(argc == 3){
        mode = atoi(argv[2]);
    }

    // Read the number of vertices and edges from the file
    fscanf(file, "%d %d", &numVertices, &numEdges);
   
    if (numVertices == -1 || numEdges == -1){
        printf("Wrongly formatted file\n");
        return 0;
    }
    //INIZIALIZATION

    //create graph
    struct Graph *g = (struct Graph *)malloc(sizeof(struct Graph));
    g->V = numVertices;  //total vertices
    g->E = numEdges;  //total edges

    //array of edges for graph
    g->edge = (struct Edge *)malloc(g->E * sizeof(struct Edge));

    // CREATION OF EDGES READING FROM FILE
    for (int i = 0; i < g->E; i++) {
        fscanf(file, "%d %d %d", &g->edge[i].u, &g->edge[i].v, &g->edge[i].w);
    }
    
    fclose(file);

    //start timer
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //bellman ford call
    bellmanford(g, 0, mode);
    
    //end timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsed_time, start, stop);   

    elapsed_time /= 1000;
    // Print the elapsed time
    printf("Elapsed time: %f s\n", elapsed_time);
    
    // Clean up resources
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //WRITE RESULTS TO A FILE

    char *last_word;

    // Find the last occurrence of '\' in the file path
    last_word = strrchr(argv[1], '/');
    int length = strlen(last_word);
    for (int i = 0; i < length; i++) {
        last_word[i] = last_word[i + 1];
    }


    char file_name[256] = "CUDA/resultsCUDA/result_";
    strcat(file_name, last_word);

    // Open the file in append mode
    FILE *output_file = fopen(file_name, "a");
    
    if (output_file == NULL) {
        printf("Error opening file.\n");
        return 0;
    }

    // Write results to the file
    if (mode == 0){
        fprintf(output_file, "Parallel mode,Block size: %d, Elapsed_time: %f\n", BLK_DIM, elapsed_time);
    }
    else{
        fprintf(output_file, "Sequential mode, Elapsed_time: %f\n", elapsed_time);
    }

    // Close the file
    fclose(output_file);

    return 0;
}
